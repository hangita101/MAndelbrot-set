#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "complex.cpp"
#include <fstream>
#include <iostream>
#define vptr (void **)

struct Point
{
    float x;
    float y;
    __device__ Point(float x, float y)
    {
        this->x = x;
        this->y = y;
    }
    __device__ Point()
    {
        this->x = 0;
        this->y = 0;
    }
};

__device__ Point LinearInterpolation(int x, int y, int width, int height, Point p1, Point p2)
{
    Point result;
    result.x = p1.x + ((p2.x - p1.x) / width) * x;
    result.y = p1.y + ((p2.y - p1.y) / height) * y;
    return result;
}

__device__ int mandel(Complex c, int maxIter)
{
    Complex z(0, 0);
    int i = 0;
    while (i < maxIter)
    {
        z = z * z + c;
        if (z.real * z.real + z.imag * z.imag > 4.0)
        {
            break;
        }
        i++;
    }
    return i;
}

__global__ void kernel(unsigned int *image_d, const unsigned int width, const unsigned int height)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    Point p1(-2, 2);
    Point p2(2, -2);

    if (idx < width && idy < height)
    {
        Point p = LinearInterpolation(idx, idy, width, height, p1, p2);
        image_d[idy * width + idx] = mandel(Complex(p.x, p.y), 100);
    }
}

void mandelbrot_gpu(unsigned int *image, const unsigned int width, const unsigned int height)
{
    unsigned int *image_d;
    hipMalloc(vptr(&image_d), sizeof(unsigned int) * width * height);

    dim3 noOfThreads(32, 32);
    dim3 noOfblocks((width + noOfThreads.x - 1) / noOfThreads.x, (height + noOfThreads.y - 1) / noOfThreads.y);
    kernel<<<noOfblocks, noOfThreads>>>(image_d, width, height);
    hipMemcpy(image, image_d, sizeof(unsigned int) * width * height, hipMemcpyDeviceToHost);
    hipFree(image_d);
}

int main()
{
    unsigned int width = 10500;
    unsigned int height = 10500;

    unsigned int *image = new unsigned int[width * height];

    mandelbrot_gpu(image, width, height);

    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            if (image[i * width + j] > 90)
            {
                std::cout << "@";
            }
            else if (image[i * width + j] > 80)
            {
                std::cout << "#";
            }
            else if (image[i * width + j] > 50)
            {
                std::cout << "%";
            }
            else if(image[i*width+j]>10){
                std::cout<<".";
            }
            else
            {
                std::cout << " ";
            }
            // std::cout<<image[i*width+j]<<" ";
        }
        std::cout << "\n";
    }

    delete[] image;
}